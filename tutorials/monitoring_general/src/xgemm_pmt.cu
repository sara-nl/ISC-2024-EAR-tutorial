#include "hip/hip_runtime.h"
#include <stdio.h> // needed for ‘printf’ 
#include <stdlib.h> // needed for ‘RAND_MAX’ 
#include <omp.h> // needed for OpenMP 
#include <time.h> // needed for clock() and CLOCKS_PER_SEC etc
#include "helper.h" // local helper header to clean up code
#include <pmt.h> // needed for PMT
#include <pmt/Rapl.h> // needed for RAPL
#include<iostream> // needed for CPP IO ... cout, endl etc etc


#ifdef USE_DOUBLE
typedef double X_TYPE;
#else
typedef float X_TYPE;
#endif

void initialize_matrices(X_TYPE* A, X_TYPE* B, X_TYPE* C, int ROWS, int COLUMNS){
    // Do this in Parallel with OpenMP
    // Needs a seperate seed per thread as rand() is obtaining a mutex and therefore locking each thread.
    unsigned int globalSeed = clock();  
    #pragma omp parallel for
    for (int i = 0; i < ROWS * COLUMNS; i++)
        {
          unsigned int randomState = i ^ globalSeed;
          A[i] = (X_TYPE) rand_r(&randomState) / RAND_MAX;
          B[i] = (X_TYPE) rand_r(&randomState) / RAND_MAX;
          C[i] = 0.0 ;
        }
}

__global__ void simple_matrix_multiply(X_TYPE* D_A, X_TYPE* D_B, X_TYPE* D_C, int ROWS, int COLUMNS){
    
    int local_COLUMN = threadIdx.x + blockIdx.x * blockDim.x;
		int local_ROW = threadIdx.y + blockIdx.y * blockDim.y;
		int local_index = local_COLUMN + local_ROW * ROWS; // Right now this only works for symetric matricies
		int tmp = 0;  
    
    if(local_ROW < ROWS && local_COLUMN < COLUMNS){
			for(int k=0; k<COLUMNS; k++){
				tmp += D_A[local_ROW * ROWS + k] * D_B[k * COLUMNS + local_COLUMN];
			}
			D_C[local_index] = tmp;
		}
}


int main( int argc, char *argv[] )  {

    printf("X_TYPE size is (%d) bytes \n",sizeof (X_TYPE));


  int ROWS;
  int COLUMNS;
  int N;

  /* DUMB bools needed for the argument parsing logic */
  bool openmp = false;
  bool simple = true;
  bool sanity_check = false;
  
  /* VERY DUMB Argument Parsers */
  N = parse_arguments(argc, argv, &simple, &openmp, &sanity_check);
  ROWS = N;
  COLUMNS = N;

  /* declare the arrays...  better to do it as 1D arrays for CUDA */

  // First allocated them on the host (CPU)
    X_TYPE* A = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));
    X_TYPE* B = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));
    X_TYPE* C = (X_TYPE*)malloc((ROWS * COLUMNS) * sizeof(X_TYPE));

  // Then Allocate them on the GPUs
  X_TYPE* D_A;
  X_TYPE* D_B;
  X_TYPE* D_C;
  hipMalloc((void**)&D_A, sizeof( X_TYPE ) * (ROWS * COLUMNS));
  hipMalloc((void**)&D_B, sizeof( X_TYPE ) * (ROWS * COLUMNS));
  hipMalloc((void**)&D_C, sizeof( X_TYPE ) * (ROWS * COLUMNS));

  double start = omp_get_wtime();  

  initialize_matrices(A, B, C, ROWS, COLUMNS);
    
  double end = omp_get_wtime(); 
  printf("Init TIME: %f sec\n",(end-start));

    // THIS IS NEW !!!!!!!
  auto GPUsensor = pmt::nvml::NVML::Create();
  auto CPUsensor = pmt::rapl::Rapl::Create();

  /*======================================================================*/
  /*                START of Section of the code that matters!!!          */
  /*======================================================================*/

  /* Simple matrix multiplication */
  /*==============================*/
    int block_size = 512;
    int grid_size = ((ROWS + block_size) / block_size);
    
    //Start the PMT "sensor"
    auto GPUstart = GPUsensor->Read(); // READING the GPU via NVML
    auto CPUstart = CPUsensor->Read(); // READING the CPU via RAPL

    // Transfer data from host to device memory
    hipMemcpy(D_A, A, sizeof(X_TYPE) * (ROWS * COLUMNS), hipMemcpyHostToDevice);
    hipMemcpy(D_B, B, sizeof(X_TYPE) * (ROWS * COLUMNS), hipMemcpyHostToDevice);
    
    simple_matrix_multiply<<<grid_size,block_size>>>(D_A, D_B, D_C, ROWS, COLUMNS);

   // Transfer data from device to host memory
    hipMemcpy(C, D_C, sizeof(X_TYPE) * (ROWS * COLUMNS), hipMemcpyDeviceToHost);


    //Start the PMT "sensor"
    auto GPUend = GPUsensor->Read();
    auto CPUend = CPUsensor->Read();

    std::cout << "SIZE: " << N << std::endl;
    std::cout << "(RAPL) CPU_TIME: " << pmt::PMT::seconds(CPUstart, CPUend) << " | (NVML) GPU_TIME: " << pmt::PMT::seconds(GPUstart, GPUend) << " s"<< std::endl;
    std::cout << "(RAPL) CPU_JOULES: " << pmt::PMT::joules(CPUstart, CPUend) << " | (NVML) GPU_JOULES: " << pmt::PMT::joules(GPUstart, GPUend) << " J"<< std::endl;
    std::cout << "(RAPL) CPU_WATTS: " << pmt::PMT::watts(CPUstart, CPUend) << " | (NVML) GPU_WATTS: " << pmt::PMT::watts(GPUstart, GPUend) << " W"<< std::endl;
    std::cout << "Total TIME: " << (pmt::PMT::seconds(CPUstart, CPUend) + pmt::PMT::seconds(GPUstart, GPUend))*0.5 << " s"<< std::endl;
    std::cout << "Total JOULES: " << (pmt::PMT::joules(CPUstart, CPUend) + pmt::PMT::joules(GPUstart, GPUend)) << " J"<< std::endl;
    std::cout << "Total WATTS: " << (pmt::PMT::watts(CPUstart, CPUend) + pmt::PMT::watts(GPUstart, GPUend)) << " W"<< std::endl;
    
  /*======================================================================*/
  /*                 END of Section of the code that matters!!!           */
  /*======================================================================*/

 // Deallocate device memory
    hipFree(D_A);
    hipFree(D_B);
    hipFree(D_C);

  // Deallocate host memory
  free(A);
  free(B);
  free(C);
}
