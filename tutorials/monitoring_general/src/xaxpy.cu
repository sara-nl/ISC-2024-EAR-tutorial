#include "hip/hip_runtime.h"
#include <stdio.h> // needed for ‘printf’ 
#include <omp.h> // needed for OpenMP 
#include <time.h> // needed for clock() and CLOCKS_PER_SEC etc
#include "helper.h" // local helper header to clean up code

#ifdef USE_DOUBLE
typedef double X_TYPE;
#else
typedef float X_TYPE;
#endif

void simple_axpy(int n, X_TYPE a, X_TYPE * x, X_TYPE * y){

    printf("(Simple) saxpy of Array of size (%d)\n",n);

    for(int i=0; i<n; i++){
        y[i] = a * x[i] + y[i];
    }
}

void openmp_axpy(int n, X_TYPE a, X_TYPE * x, X_TYPE * y){
    int num_threads = omp_get_max_threads();

    printf("(OpenMP) saxpy of Array of size (%d)\n",n);
    printf("Using %d Threads\n", num_threads);
    #pragma omp parallel for
    for(int i=0; i<n; i++){
        y[i] = a * x[i] + y[i];
    }
}

__global__ void gpu_axpy(int n, X_TYPE a, X_TYPE * x, X_TYPE * y) {
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handling arbitrary vector size
    if (tid < n){
        y[tid] = a * x[tid] + y[tid];
    }
}





int main( int argc, char *argv[] )  {

    int N;
    /* DUMB bools needed for the argument parsing logic */
    bool openmp = false;
    bool simple = true;
    bool sanity_check = false;
    
    /* VERY DUMB Argument Parsers */
    N = parse_arguments(argc, argv, &simple, &openmp, &sanity_check);

    X_TYPE *d_sx; /* n is an array of N integers */
    X_TYPE *d_sy; /* n is an array of N integers */

    X_TYPE a = 2.0;
    // Allocate Host memory 
    X_TYPE* sx = (X_TYPE*)malloc(N * sizeof(X_TYPE));
    X_TYPE* sy = (X_TYPE*)malloc(N * sizeof(X_TYPE));


    // Allocate device memory 
    hipMalloc((void**)&d_sx, sizeof(X_TYPE) * N);
    hipMalloc((void**)&d_sy, sizeof(X_TYPE) * N);
    // hipMalloc((void**)&d_a, sizeof(X_TYPE));

    printf("X_TYPE size is (%d)\n",sizeof (X_TYPE));
    /* Simple saxpy */
    /*==============================*/
    if (true == simple)
    {

        int block_size = 512;
        int grid_size = ((N + block_size) / block_size);
      clock_t t; // declare clock_t (long type)
      t = clock(); // start the clock
    
        // Transfer data from host to device memory
        hipMemcpy(d_sx, sx, sizeof(X_TYPE) * N, hipMemcpyHostToDevice);
        hipMemcpy(d_sy, sy, sizeof(X_TYPE) * N, hipMemcpyHostToDevice);
        //hipMemcpy(d_a, a, sizeof(X_TYPE) , hipMemcpyHostToDevice);

        gpu_axpy<<<grid_size,block_size>>>(N, a, d_sx, d_sy);

        hipMemcpy(sy, d_sy, sizeof(X_TYPE) * N, hipMemcpyDeviceToHost);
    
      t = clock() - t; // stop the clock    
      double time_taken = ((double)t)/CLOCKS_PER_SEC; // convert to seconds (and long to double)
      printf("TIME: %f s\n",time_taken);
    }

    /* OpenMP parallel saxpy */
    /*==============================*/
    if (true == openmp)
    {

    // omp_get_wtime needed here because clock will sum up time for all threads
    double start = omp_get_wtime();  

    openmp_axpy(N, 2.0, sx, sy);
    
    double end = omp_get_wtime(); 
    printf("TIME: %f s\n",(end-start));

    }


}
