#include "hip/hip_runtime.h"
#include <stdio.h> // needed for ‘printf’ 
#include <omp.h> // needed for OpenMP 
#include <time.h> // needed for clock() and CLOCKS_PER_SEC etc
#include "helper.h" // local helper header to clean up code
#include <pmt.h> // needed for PMT
#include <pmt/Rapl.h> // needed for RAPL
#include <iostream> // needed for CPP IO ... cout, endl etc etc

#ifdef USE_DOUBLE
typedef double X_TYPE;
#else
typedef float X_TYPE;
#endif

void simple_axpy(int n, X_TYPE a, X_TYPE * x, X_TYPE * y){

    printf("(Simple) saxpy of Array of size (%d)\n",n);

    for(int i=0; i<n; i++){
        y[i] = a * x[i] + y[i];
    }
}

void openmp_axpy(int n, X_TYPE a, X_TYPE * x, X_TYPE * y){
    int num_threads = omp_get_max_threads();

    printf("(OpenMP) saxpy of Array of size (%d)\n",n);
    printf("Using %d Threads\n", num_threads);
    #pragma omp parallel for
    for(int i=0; i<n; i++){
        y[i] = a * x[i] + y[i];
    }
}

__global__ void gpu_axpy(int n, X_TYPE a, X_TYPE * x, X_TYPE * y) {
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handling arbitrary vector size
    if (tid < n){
        y[tid] = a * x[tid] + y[tid];
    }
}





int main( int argc, char *argv[] )  {

    printf("X_TYPE size is (%d) bytes \n",sizeof (X_TYPE));

    int N;
    /* DUMB bools needed for the argument parsing logic */
    bool openmp = false;
    bool simple = true;
    bool sanity_check = false;
    
    /* VERY DUMB Argument Parsers */
    N = parse_arguments(argc, argv, &simple, &openmp, &sanity_check);

    X_TYPE *d_sx; /* n is an array of N integers */
    X_TYPE *d_sy; /* n is an array of N integers */

    X_TYPE a = 2.0;
    // Allocate Host memory 
    X_TYPE* sx = (X_TYPE*)malloc(N * sizeof(X_TYPE));
    X_TYPE* sy = (X_TYPE*)malloc(N * sizeof(X_TYPE));

    // Allocate device memory 
    hipMalloc((void**)&d_sx, sizeof(X_TYPE) * N);
    hipMalloc((void**)&d_sy, sizeof(X_TYPE) * N);

    // THIS IS NEW !!!!!!!
    auto GPUsensor = pmt::nvml::NVML::Create();
    auto CPUsensor = pmt::rapl::Rapl::Create();

    /* Simple saxpy */
    /*==============================*/
    if (true == simple)
    {

        int block_size = 512;
        int grid_size = ((N + block_size) / block_size);
        
        //Start the PMT "sensor"
        auto GPUstart = GPUsensor->Read(); // READING the GPU via NVML
        auto CPUstart = CPUsensor->Read(); // READING the CPU via RAPL
        
        // Transfer data from host to device memory
        hipMemcpy(d_sx, sx, sizeof(X_TYPE) * N, hipMemcpyHostToDevice);
        hipMemcpy(d_sy, sy, sizeof(X_TYPE) * N, hipMemcpyHostToDevice);

        gpu_axpy<<<grid_size,block_size>>>(N, a, d_sx, d_sy);

        hipMemcpy(sy, d_sy, sizeof(X_TYPE) * N, hipMemcpyDeviceToHost);

        //Start the PMT "sensor"
        auto GPUend = GPUsensor->Read();
        auto CPUend = CPUsensor->Read();

        std::cout << "SIZE: " << N << std::endl;
        std::cout << "(RAPL) CPU_TIME: " << pmt::PMT::seconds(CPUstart, CPUend) << " | (NVML) GPU_TIME: " << pmt::PMT::seconds(GPUstart, GPUend) << " s"<< std::endl;
        std::cout << "(RAPL) CPU_JOULES: " << pmt::PMT::joules(CPUstart, CPUend) << " | (NVML) GPU_JOULES: " << pmt::PMT::joules(GPUstart, GPUend) << " J"<< std::endl;
        std::cout << "(RAPL) CPU_WATTS: " << pmt::PMT::watts(CPUstart, CPUend) << " | (NVML) GPU_WATTS: " << pmt::PMT::watts(GPUstart, GPUend) << " W"<< std::endl;
        std::cout << "Total TIME: " << (pmt::PMT::seconds(CPUstart, CPUend) + pmt::PMT::seconds(GPUstart, GPUend))*0.5 << " s"<< std::endl;
        std::cout << "Total JOULES: " << (pmt::PMT::joules(CPUstart, CPUend) + pmt::PMT::joules(GPUstart, GPUend)) << " J"<< std::endl;
        std::cout << "Total WATTS: " << (pmt::PMT::watts(CPUstart, CPUend) + pmt::PMT::watts(GPUstart, GPUend)) << " W"<< std::endl;
    
    }
    /* OpenMP parallel saxpy */
    /*==============================*/
    if (true == openmp)
    {

    // omp_get_wtime needed here because clock will sum up time for all threads
    double start = omp_get_wtime();  

    openmp_axpy(N, 2.0, sx, sy);
    
    double end = omp_get_wtime(); 
    printf("TIME: %f sec\n",(end-start));

    }


}
